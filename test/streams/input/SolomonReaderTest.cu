#include "hip/hip_runtime.h"
#include <catch/catch.hpp>

#include "config.hpp"

#include "algorithms/CartesianDistance.cu"
#include "models/Problem.hpp"
#include "models/Resources.hpp"
#include "streams/input/SolomonReader.cu"

#include "test_utils/VectorUtils.hpp"

#include <fstream>

using namespace vrp::algorithms;
using namespace vrp::models;
using namespace vrp::streams;

SCENARIO("Can create customers data.", "[streams][solomon][T1]") {
  std::fstream input(SOLOMON_TESTS_PATH "T1.txt");

  auto problem = SolomonReader<CartesianDistance>::read(input);

  CHECK_THAT(vrp::test::copy(problem.customers.demands),
             Catch::Matchers::Equals(std::vector<int>{0, 1, 1, 1}));
  CHECK_THAT(vrp::test::copy(problem.customers.services),
            Catch::Matchers::Equals(std::vector<int>(4, 1)));
  CHECK_THAT(vrp::test::copy(problem.customers.starts),
             Catch::Matchers::Equals(std::vector<int>(4, 0)));
  CHECK_THAT(vrp::test::copy(problem.customers.ends),
             Catch::Matchers::Equals(std::vector<int>(4, 1000)));
}

SCENARIO("Can create routing data.", "[streams][solomon][T1]") {
  std::fstream input(SOLOMON_TESTS_PATH "T1.txt");

  auto problem = SolomonReader<CartesianDistance>::read(input);

  CHECK_THAT(vrp::test::copy(problem.routing.distances),
             Catch::Matchers::Equals(std::vector<float>{0, 1, 3, 7, 1, 0, 2, 6, 3, 2, 0, 4, 7, 6, 4, 0}));
  CHECK_THAT(vrp::test::copy(problem.routing.durations),
             Catch::Matchers::Equals(std::vector<int>{0, 1, 3, 7, 1, 0, 2, 6, 3, 2, 0, 4, 7, 6, 4, 0}));
}

SCENARIO("Can create resources data.", "[streams][solomon][T1]") {
  std::fstream input(SOLOMON_TESTS_PATH "T1.txt");

  auto problem = SolomonReader<CartesianDistance>::read(input);

  CHECK_THAT(vrp::test::copy(problem.resources.capacities),
             Catch::Matchers::Equals(std::vector<int>{ 10 }));
  CHECK_THAT(vrp::test::copy(problem.resources.distanceCosts),
             Catch::Matchers::Equals(std::vector<float>{ 1 }));
  CHECK_THAT(vrp::test::copy(problem.resources.timeCosts),
             Catch::Matchers::Equals(std::vector<float>{ 0 }));
  CHECK_THAT(vrp::test::copy(problem.resources.waitingCosts),
             Catch::Matchers::Equals(std::vector<float>{ 0 }));
  CHECK_THAT(vrp::test::copy(problem.resources.timeLimits),
             Catch::Matchers::Equals(std::vector<int>{ std::numeric_limits<int>::max() }));
}