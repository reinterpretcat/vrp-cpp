#include "hip/hip_runtime.h"
#include "algorithms/Distances.cu"
#include "streams/input/SolomonReader.cu"
#include "streams/output/GeoJsonWriter.cu"
#include "heuristics/NearestNeighbor.hpp"
#include "solver/genetic/Populations.hpp"

#include <algorithm>
#include <fstream>
#include <utility>

using namespace vrp::algorithms;
using namespace vrp::heuristics;
using namespace vrp::genetic;
using namespace vrp::streams;

namespace {

using IntCoord = std::pair<int,int>;
using GeoCoord = std::pair<double,double>;

/// Maps int coordinate as double without changes.
struct DefaultMapper final {
  explicit DefaultMapper(double scale) : scale(scale) {}

  GeoCoord operator()(const std::pair<IntCoord,IntCoord> intBoundingBox,
                      const GeoCoord &coordinate) const {
    return std::make_pair(coordinate.first * scale, coordinate.second * scale);
  }
 private:
  double scale;
};

///// Maps int coordinate as geo coordinate inside the bounding box.
struct BoundingBoxMapper final {
  explicit BoundingBoxMapper(const std::pair<GeoCoord,GeoCoord> &boundingBox) :
    geoBoundingBox(geoBoundingBox) {}

  GeoCoord operator()(const std::pair<IntCoord,IntCoord> intBoundingBox,
                      const GeoCoord &coordinate) const {

    double ratioX = (coordinate.first - intBoundingBox.first.first) /
        static_cast<double>(intBoundingBox.second.first - intBoundingBox.first.first);

    double ratioY = (coordinate.second- intBoundingBox.first.second) /
        static_cast<double>(intBoundingBox.second.second - intBoundingBox.first.second);

    return std::make_pair(
        geoBoundingBox.first.first + (geoBoundingBox.second.first - geoBoundingBox.first.first) * ratioX,
        geoBoundingBox.first.second + (geoBoundingBox.second.second - geoBoundingBox.first.second) * ratioY);
  }
 private:
  std::pair<GeoCoord,GeoCoord> geoBoundingBox;
};

/// Resolves locations as geo coordinates.
template <typename Mapper = DefaultMapper>
struct LocationResolver final {
  explicit LocationResolver(std::fstream &in, const Mapper &mapper) : mapper(mapper) {
    initLocations(in);
    intBoundingBox = getBoundingBox();
  }

  GeoCoord operator()(int customer) const {
    return mapper(intBoundingBox, locations.at(static_cast<unsigned long>(customer)));
  }

 private:
  void initLocations(std::fstream &in) {
    in.clear();
    in.seekg(0, std::ios::beg);

    for (int i = 0; i < 10; ++i)
      in.ignore(std::numeric_limits<std::streamsize>::max(), in.widen('\n'));

    int id;
    std::pair<int, int> location;
    while (in) {
      in >> id >> location.first >> location.second;
      locations.emplace_back(location);
    }
  }

  std::pair<IntCoord,IntCoord> getBoundingBox() const {
    auto minMaxX = std::minmax_element(locations.begin(), locations.end(),
                                       [](const IntCoord &left, const IntCoord &right) {
                                         return left.first < right.first;
                                       });
    auto minMaxY = std::minmax_element(locations.begin(), locations.end(),
                                       [](const IntCoord &left, const IntCoord &right) {
                                         return left.second < right.second;
                                       });

    return std::make_pair(
        IntCoord {minMaxX.first->first, minMaxY.first->second },
        IntCoord {minMaxX.second->first, minMaxY.second->second });
  };

  const Mapper &mapper;
  std::vector<IntCoord> locations;
  std::pair<IntCoord,IntCoord> intBoundingBox;
};

};

int main(int argc, char* argv[]) {
  if (argc != 3)
    throw std::invalid_argument("Missing input or output argument.");

  std::fstream in(argv[1]);
  std::fstream out(argv[2]);

  auto problem = SolomonReader<geographic_distance<>>().read(in);
  auto solution = create_population<NearestNeighbor>(problem)({ 1 });

  auto mapper = DefaultMapper(1.);
  auto resolver = LocationResolver<decltype(mapper)>(in, mapper);
  GeoJsonWriter<decltype(resolver)>().write(out, solution, resolver);

  return 0;
}
