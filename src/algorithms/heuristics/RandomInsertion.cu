#include "hip/hip_runtime.h"
#include "algorithms/heuristics/RandomInsertion.hpp"
#include "algorithms/transitions/Executors.hpp"
#include "iterators/Aggregates.hpp"
#include "runtime/UniquePointer.hpp"

#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_int_distribution.h>

using namespace vrp::algorithms::heuristics;
using namespace vrp::models;
using namespace vrp::runtime;

namespace {

/// Specifies vehicle range: start, end, id, and some extra.
using VehicleRange = thrust::tuple<int, int, int, int>;

/// Represents search context: data passed through main operators.
struct SearchContext final {
  Context context;
  int base;
  int last;
  int customer;
};

/// Stores data used to estimate insertion.
struct InsertionData final {
  /// Task where vehicle range starts.
  int from;
  /// Task where vehicle range ends.
  int to;
  /// Used vehicle.
  int vehicle;
  /// Customer to be inserted
  int customer;
};

/// Stores insertion result.
struct InsertionResult final {
  /// Contains insertion data.
  InsertionData data;
  /// Insertion Point
  int point;
  /// Estimated insertion cost
  float cost;
};

EXEC_UNIT
inline InsertionResult create_invalid_data() { return {{}, -1, __FLT_MAX__}; }

/// Finds next random customer to serve.
struct find_random_customer final {
  EXEC_UNIT explicit find_random_customer(const Tasks::Shadow tasks, int base) :
    tasks(tasks), base(base), maxCustomer(tasks.customers - 1), dist(1, maxCustomer), rng() {}

  EXEC_UNIT int operator()() {
    auto start = dist(rng);
    auto customer = start;
    bool increment = start % 2 == 0;

    do {
      Plan plan = tasks.plan[base + customer];

      if (!plan.isAssigned()) return customer;

      // try to find next customer
      if (increment)
        customer = customer == maxCustomer ? 1 : customer + 1;
      else
        customer = customer == 0 ? maxCustomer : customer - 1;
    } while (customer != start);

    return -1;
  }

private:
  const Tasks::Shadow tasks;
  int base;
  int maxCustomer;
  thrust::uniform_int_distribution<int> dist;
  thrust::minstd_rand rng;
};

template<typename TransitionOp>
struct state_processor final {
  const SearchContext search;
  const TransitionOp transitionOp;

  Transition::State state;
  float cost;

  /// Restores state before insertion point.
  EXEC_UNIT void restore(int point, int base, int vehicle) {
    const auto& context = search.context;

    auto index = base + point;

    int capacity = index == 0 ? static_cast<int>(context.problem.resources.capacities[vehicle])
                              : context.tasks.capacities[index];

    int time = index == 0 ? 0 : context.tasks.times[index];

    cost = index == 0 ? 0 : context.tasks.costs[index];

    state.customer = context.tasks.ids[index];
    state.capacity = capacity;
    state.time = time;
  }

  /// Updates state within new customer.
  EXEC_UNIT Transition update(int id, int task, int base, int vehicle) {
    auto customer = variant<int, Convolution>::create(id);
    auto details = Transition::Details{base, task, task + 1, customer, vehicle};
    Transition transition = transitionOp.create(details, state);

    if (!transition.isValid()) return transition;

    cost += transitionOp.estimate(transition);

    state.customer = transition.details.customer.get<int>();
    state.time += transition.delta.duration();
    state.capacity -= transition.delta.demand;

    return transition;
  }

  EXEC_UNIT int customer(int task) { return search.context.tasks.ids[task]; }

  EXEC_UNIT float costs(int task) { return search.context.tasks.costs[task]; }
};

/// Estimates insertion to a given arc.
template<typename TransitionOp>
struct estimate_insertion final {
  const InsertionData data;
  state_processor<TransitionOp> stateOp;
  int base;

  /// @param task Task index from which arc starts.
  EXEC_UNIT InsertionResult operator()(int point) {
    int vehicle = data.vehicle;
    float cost = stateOp.costs(base + data.to);

    stateOp.restore(point, base, vehicle);

    if (!stateOp.update(data.customer, point, base, vehicle).isValid())
      return create_invalid_data();

    for (int i = point + 1; i <= data.to; ++i) {
      auto customer = stateOp.customer(base + i);
      if (!stateOp.update(customer, i, base, vehicle).isValid()) return create_invalid_data();
    }

    return InsertionResult{
      {data.from, data.to, vehicle, data.customer}, point, stateOp.cost - cost};
  }
};

///// Compares two arcs using their insertion costs.
struct compare_arcs_value final {
  EXEC_UNIT InsertionResult operator()(const InsertionResult& left,
                                       const InsertionResult& right) const {
    return left.cost < right.cost ? left : right;
  }
};

///// Compares two arcs using their insertion costs.
struct compare_arcs_logical final {
  EXEC_UNIT bool operator()(const InsertionResult& left, const InsertionResult& right) const {
    return left.cost < right.cost;
  }
};

/// Finds the "best" arc from single tour where given customer can be inserted.
template<typename TransitionOp>
struct find_best_arc final {
  const SearchContext search;
  const TransitionOp transitionOp;
  vector_ptr<InsertionResult> results;

  EXEC_UNIT InsertionResult operator()(const VehicleRange& range) const {
    if (thrust::get<0>(range) == -1) return create_invalid_data();

    int from = thrust::get<0>(range);
    int to = thrust::get<1>(range);
    int vehicle = thrust::get<2>(range);

    auto data = InsertionData{from, to, vehicle, search.customer};

    results[vehicle] = thrust::transform_reduce(
      exec_unit_policy{}, thrust::make_counting_iterator(from),
      thrust::make_counting_iterator(to + 1),

      estimate_insertion<TransitionOp>{
        data, state_processor<TransitionOp>{search, transitionOp, {}, 0}, search.base},

      InsertionResult{data, -1, __FLT_MAX__}, compare_arcs_value{});

    return {};
  }
};

/// Represents operator which helps to create vehicle ranges without extra memory footprint.
struct create_vehicle_ranges final {
  EXEC_UNIT VehicleRange operator()(const VehicleRange& left, const VehicleRange& right) {
    auto leftStart = thrust::get<0>(left);
    auto leftEnd = thrust::get<1>(left);
    auto leftVehicle = thrust::get<2>(left);
    auto leftExtra = thrust::get<3>(left);

    auto rightStart = thrust::get<0>(right);
    auto rightEnd = thrust::get<1>(right);
    auto rightVehicle = thrust::get<2>(right);

    if (rightStart == 0) return {1, leftExtra != -1 ? 1 : leftEnd, 0, -1};

    if (leftExtra != -1) {
      // continue with this vehicle
      if (leftExtra == rightVehicle) {
        return {-1, leftStart - 1, leftExtra, -1};
      }
      // vehicle was used only once
      else {
        return {leftStart - 1, leftStart - 1, leftExtra, rightVehicle};
      }
    }

    if (leftVehicle != rightVehicle) return {rightStart + 1, leftEnd, leftVehicle, rightVehicle};

    return {-1, leftEnd, leftVehicle, -1};
  }
};

/// Finds the "best" insertion point for given customer inside all tours.
template<typename TransitionOp>
struct find_insertion_point final {
  const TransitionOp transitionOp;
  unique_ptr<vector_ptr<InsertionResult>> results;

  /// @returns Task index from which to perform transition.
  EXEC_UNIT InsertionResult operator()(const SearchContext& search, int vehicle) {
    auto iterator = thrust::make_zip_iterator(
      thrust::make_tuple(thrust::make_counting_iterator(0), thrust::make_constant_iterator(0),
                         search.context.tasks.vehicles, thrust::make_constant_iterator(0)));
    auto lastVehicle = search.context.tasks.vehicles[search.base + search.last - 1];

    // first customer in tour
    if (search.last == 1 || lastVehicle != vehicle)
      return InsertionResult{{0, search.last, vehicle, search.customer}, search.last, 0};

    thrust::exclusive_scan(
      exec_unit_policy{},

      thrust::make_zip_iterator(thrust::make_tuple(
        thrust::make_reverse_iterator(thrust::make_counting_iterator(search.last)),
        thrust::make_constant_iterator(-1),
        thrust::make_reverse_iterator(search.context.tasks.vehicles + search.base + search.last),
        thrust::make_constant_iterator(-1))),

      thrust::make_zip_iterator(thrust::make_tuple(
        thrust::make_reverse_iterator(thrust::make_counting_iterator(-1)),
        thrust::make_constant_iterator(1),
        thrust::make_reverse_iterator(search.context.tasks.vehicles + search.base),
        thrust::make_constant_iterator(-1))),

      vrp::iterators::make_aggregate_output_iterator(
        iterator, find_best_arc<TransitionOp>{search, transitionOp, *results.get()}),

      VehicleRange{-1, search.last - 1, lastVehicle, -1},

      create_vehicle_ranges{});

    return *thrust::min_element(exec_unit_policy{}, *results.get(),
                                *results.get() + lastVehicle + 1, compare_arcs_logical{});
  }
};

///// Inserts a new customer in between existing ones.
template<typename TransitionOp>
struct insert_customer final {
  const TransitionOp transitionOp;

  /// @returns Index of last task.
  EXEC_UNIT int operator()(const SearchContext& search, const InsertionResult& data) {
    return data.point == search.last ? insertLast(search, data) : insertInBetween(search, data);
  }

private:
  /// Inserts new customer as last.
  EXEC_UNIT int insertLast(const SearchContext& search, const InsertionResult& result) {
    auto customer = variant<int, Convolution>::create(result.data.customer);

    auto details = Transition::Details{search.base, result.data.from, result.data.to, customer,
                                       result.data.vehicle};
    auto transition = transitionOp.create(details);
    auto cost = transitionOp.estimate(transition);
    return transitionOp.perform(transition, cost);
  }

  /// Inserts new customer in single tour.
  EXEC_UNIT int insertInBetween(const SearchContext& search, const InsertionResult& result) {
    int begin = search.base + result.point;
    int end = search.base + search.last;
    auto tasks = search.context.tasks;

    // shift everything to the right
    shift(tasks.ids + begin, tasks.ids + end);
    shift(tasks.costs + begin, tasks.costs + end);
    shift(tasks.vehicles + begin, tasks.vehicles + end);
    shift(tasks.capacities + begin, tasks.capacities + end);
    shift(tasks.times + begin, tasks.times + end);

    // insert new customer
    auto stateOp = state_processor<TransitionOp>{search, transitionOp};
    stateOp.restore(result.point, search.base, result.data.vehicle);

    // insert and recalculate affected tour
    auto last = -1;
    for (int i = result.point; i <= result.data.to; ++i) {
      auto customer =
        i == result.point ? result.data.customer : stateOp.customer(search.base + i + 1);
      auto transition = stateOp.update(customer, i, search.base, result.data.vehicle);
      auto cost = transitionOp.estimate(transition);
      last = transitionOp.perform(transition, cost);
    }

    return thrust::max(last, search.last);
  }

  /// Shifts to the right all data.
  template<typename T>
  EXEC_UNIT void shift(T begin, T end) {
    for (auto iter = end - 1; iter >= begin; --iter) {
      *(iter + 1) = *iter;
    }
  }
};

}  // namespace

namespace vrp {
namespace algorithms {
namespace heuristics {

template<typename TransitionOp>
EXEC_UNIT void random_insertion<TransitionOp>::operator()(const Context& context,
                                                          int index,
                                                          int shift) {
  const auto begin = index * context.problem.size;

  auto transitionOp = TransitionOp(context.problem, context.tasks);
  auto findCustomer = find_random_customer(context.tasks, begin);
  auto findPoint = find_insertion_point<TransitionOp>{
    transitionOp, make_unique_ptr_data<InsertionResult>(context.problem.size)};
  auto insertCustomer = insert_customer<TransitionOp>{transitionOp};

  int to = shift == 0 ? 1 : shift + 1;
  int customer = 0;
  int vehicle = context.tasks.vehicles[to - 1];

  while (to < context.problem.size) {
    customer = customer != 0 ? customer : findCustomer();

    auto search = SearchContext{context, begin, to, customer};
    auto insertion = findPoint(search, vehicle);

    // allocate new vehicle if estimation fails to insert customer
    if (insertion.point == -1) {
      ++vehicle;
      continue;
    }

    to = insertCustomer(search, insertion) + 1;

    customer = 0;
  }
}

/// NOTE make linker happy.
template class random_insertion<TransitionOperator>;

}  // namespace heuristics
}  // namespace algorithms
}  // namespace vrp
