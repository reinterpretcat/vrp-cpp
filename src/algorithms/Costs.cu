#include "hip/hip_runtime.h"
#ifndef VRP_ALGORITHMS_COSTS_HPP
#define VRP_ALGORITHMS_COSTS_HPP

#include "algorithms/Transitions.cu"
#include "models/Problem.hpp"
#include "models/Resources.hpp"
#include "models/Tasks.hpp"
#include "models/Transition.hpp"
#include "utils/Memory.hpp"

#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/unique.h>

namespace vrp {
namespace algorithms {

/// Calculates cost of transition.
struct calculate_transition_cost final {
  const vrp::models::Resources::Shadow resources;

  __host__ __device__
  explicit calculate_transition_cost(const vrp::models::Resources::Shadow &resources) :
      resources(resources) {}

  __host__ __device__
  float operator()(const vrp::models::Transition &transition) const {
    int vehicle = transition.details.vehicle;

    auto distance = transition.delta.distance * resources.distanceCosts[vehicle];
    auto traveling = transition.delta.traveling * resources.timeCosts[vehicle];
    auto waiting = transition.delta.waiting * resources.waitingCosts[vehicle];
    auto serving = transition.delta.serving * resources.timeCosts[vehicle];

    return distance + traveling + waiting + serving;
  }
};

/// Calculates total cost of solution.
struct calculate_total_cost final {
  /// Represents cost mode used in cost aggregator.
  struct CostModel final {
    float total;
    vrp::models::Problem::Shadow problem;
    vrp::models::Tasks::Shadow tasks;
  };

  /// Aggregates all costs.
  struct aggregate_cost final {
    CostModel *costModel;
    int lastCustomer;
    int baseTask;

    template<class Tuple>
    __device__
    float operator()(const Tuple &tuple) {
      const int task = lastCustomer - thrust::get<0>(tuple);
      const int vehicle = thrust::get<1>(tuple);
      const int depot = 0;
      const float cost = thrust::get<2>(tuple);

      auto details = vrp::models::Transition::Details{baseTask + task, -1, depot, vehicle};
      auto transition = create_transition(costModel->problem, costModel->tasks)(details);
      auto returnCost = calculate_transition_cost(costModel->problem.resources)(transition);
      auto routeCost = cost + returnCost;

      // NOTE to use atomicAdd, variable has to be allocated in device memory,
      // not in registers
      atomicAdd(&costModel->total, routeCost);

      return routeCost;
    }
  };

  __host__
  float operator()(const vrp::models::Problem &problem,
                   vrp::models::Tasks &tasks,
                   int solution = 0) const {
    int end = tasks.customers * (solution + 1);
    int rbegin = tasks.size() - end;
    int rend = rbegin + tasks.customers;

    auto model = vrp::utils::allocate<CostModel>({0, problem.getShadow(), tasks.getShadow()});

    thrust::unique_by_key_copy(
        thrust::device,
        tasks.vehicles.rbegin() + rbegin,
        tasks.vehicles.rbegin() + rend,
        thrust::make_zip_iterator(thrust::make_tuple(
            thrust::make_counting_iterator(0),
            tasks.vehicles.rbegin() + rbegin,
            tasks.costs.rbegin() + rbegin)
        ),
        thrust::make_discard_iterator(),
        thrust::make_transform_output_iterator(
            thrust::make_discard_iterator(),
            aggregate_cost{model.get(), tasks.customers - 1, end - tasks.customers}
        )
    );

    return vrp::utils::release(model).total;
  }
};

}
}

#endif //VRP_ALGORITHMS_COSTS_HPP
